/* ITESM QRO
 Mariana Perez Garcia A01206747
 Lab 2 */

 #include "hip/hip_runtime.h"
 #include <stdio.h>

// Device function that multiplies two matrices
 __global__ void matrixMultiplication(float* a, float* b, float* c, int r1, int c1, int r2, int c2) {
   // get row
   int row = blockIdx.y * blockDim.y + threadIdx.y;
   // get col
   int col = blockIdx.x * blockDim.x + threadIdx.x;

   float temp =  0;

   if(row < r1 && col < c2) {
     // each thread computes one element of the block
     for(int i = 0; i < c1; i++) {
       temp += a[row * c1 + i] * b[i * c2 + col];
     }
   }
   c[row * r1 + col] = temp;
 }


// Function that will fill the matrix where i is the number that will be in m[0,0]
 void fill_array_test(float* array, int rows, int cols, int i) {
   for(int j = 0; j < rows; j++) {
     for(int k = 0; k < cols; k++) {
       if (((j + k *rows) + i) >= 10) {
         array[j + k * rows] = (float)((j + k *rows) + i) - 10;
       } else {
         array[j + k * rows] = (float)((j + k *rows) + i);
       }
     }
   }
 }

// Function that displays the matrix
 void display_array(const char *text, float *array, int rows, int cols) {
   printf("%s", text);
  for(int i = 0; i < rows * cols; i++) {
    if(i % cols == 0) {
      printf("\n");
    }
    printf("%.2f ", array[i]);
  }
}

// Host function that will be in charge of sending the data to the device
int multiply(int num_test, int rows1, int cols1, int rows2, int cols2, int start1, int start2) {
  // if the matrices are not compatible return an error
  if(rows1 != cols2) {
    printf("Error: Cannot perform matrix multiplication");
    return -1;
  }
  float* a, *b, *c, *d_a, *d_b, *d_c;
  // get number of threads per block
  int tpb = rows1;
  // get number of blocks
  int numBlocks = (tpb + (rows1 * cols2)) / tpb;
  printf("Test %i: \n", num_test);

  // Allocate memory for the matrices
  a = (float*)malloc(sizeof(float) * rows1 * cols1);
  b = (float*)malloc(sizeof(float) * rows2 * cols2);
  c = (float*)malloc(sizeof(float) * rows1 * cols2);

  // fill the matrices
  fill_array_test(a, rows1, cols1, start1);
  fill_array_test(b, rows2, cols2, start2);

  // display matrices
  display_array("Matrix A:", a, rows1, cols1);
  printf("\n");
  display_array("Matrix B:", b, rows2, cols2);
  printf("\n");

  // Allocate memory in the GPU
  hipMalloc((void**)&d_a, sizeof(float) * rows1 * cols1);
  hipMalloc((void**)&d_b, sizeof(float) * rows2 * cols2);
  hipMalloc((void**)&d_c, sizeof(float) * rows1 * cols2);

  // Copy the values from CPU matrices to GPU matrices
  hipMemcpy(d_a, a, rows1 * cols1 * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, rows2 * cols2 * sizeof(float), hipMemcpyHostToDevice);

  // declare the number of blocks
  dim3 blocks(numBlocks,numBlocks);
  // declare the number of threads
  dim3 threads(tpb,tpb);

  // call GPU function
  matrixMultiplication<<<blocks, threads>>>(d_a, d_b, d_c, rows1, cols1, rows2, cols2);

  hipMemcpy(c, d_c, rows1 * cols2 * sizeof(float), hipMemcpyDeviceToHost);

  // Display resulting array
  display_array("Resulting Matrix:", c, rows1, cols2);
  printf("\n\n");

  // free CPU
  free(a);
  free(b);
  free(c);

  // free GPU
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  return 0;
}

 int main() {
   /* -------------------------------------------------------
                        TEST 1
   -------------------------------------------------------
   Test1:
   Matrix A:
   1.00 2.00
   3.00 4.00
   Matrix B:
   4.00 5.00
   6.00 7.00
   Resulting Matrix:
   16.00 19.00
   36.00 43.00 */

   /*Multiply function that takes as parameters (no_of_test_case, no_of_rows_for_matrix1,
   no_of_cols_for_matrix1, no_of_rows_for_matrix2, no_of_cols_for_matrix2, start_of_matrix1,
   start_of_matrix2) */
   multiply(1,2,2,2,2,1,4);
   /* -------------------------------------------------------
                        TEST 2
   ---------------------------------------------------------
   Test 2:
   Matrix A:
   1.00 2.00 3.00
   4.00 5.00 6.00
   Matrix B:
   6.00 7.00
   8.00 9.00
   0.00 1.00
   Resulting Matrix:
   22.00 28.00
   64.00 79.00 */

   /*Multiply function that takes as parameters (no_of_test_case, no_of_rows_for_matrix1,
   no_of_cols_for_matrix1, no_of_rows_for_matrix2, no_of_cols_for_matrix2, start_of_matrix1,
   start_of_matrix2) */
   multiply(2,2,3,3,2,1,6);

   /* -------------------------------------------------------
                        TEST 3
   ---------------------------------------------------------
   Test 3:
   Matrix A:
   1.00 2.00
   3.00 4.00
   5.00 6.00
   Matrix B:
   6.00 7.00 8.00
   9.00 0.00 1.00
   Resulting Matrix:
   24.00 7.00 10.00
   54.00 21.00 28.00
   84.00 35.00 46.00 */

   /*Multiply function that takes as parameters (no_of_test_case, no_of_rows_for_matrix1,
   no_of_cols_for_matrix1, no_of_rows_for_matrix2, no_of_cols_for_matrix2, start_of_matrix1,
   start_of_matrix2) */
   multiply(3,3,2,2,3,1,6);

   return 0;

 }
