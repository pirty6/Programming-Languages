/* ITESM QRO
 Mariana Perez Garcia A01206747
 Lab 1 */

#include "hip/hip_runtime.h"
#include <stdio.h>

#define MAX 100000 // max size
#define TPB 512 // threads per block

// Calculates PI parallely (Riemann Sum) using the GPU.
__global__ void pi(double *res, long max){
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  double width = 1.0 / max;
  int id = index;
  double mid;
  // All the values are added into their corresponding place in the res array
  while(id < max){
    mid = (id + 0.5) * width;
    res[id] = (4.0 / (1.0 + mid * mid)) * width;

    id = id + blockDim.x * gridDim.x;
  }
}

int main(){
  double *res; // cpu variable
  double *d_res; // gpu variable

  // Allocate memory of a double array in the cpu
  res = (double*) malloc(sizeof(double) * MAX); // Result Array

  // Allocate memory of a double array in the gpu
  hipMalloc((void**)&d_res, sizeof(double) * MAX);

  // Call function pi in the gpu
  pi<<< (MAX / TPB), TPB>>>(d_res, MAX);

  // Copy resulting array from gpu to cpu
  hipMemcpy(res, d_res, MAX * sizeof(double), hipMemcpyDeviceToHost);
  double sum = 0.0;

  // Add all the values in the array to the final result
  for (long i = 0; i < MAX; i++) {
    sum += res[i];
  }

  // Print result
  printf("Pi: %f\n", sum);

  // Free cpu memory
  free(res);

  // Free cpu memory
  hipFree(d_res);

  return 0;
}
